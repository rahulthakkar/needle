#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <nvfunctional>
#include <pybind11/numpy.h>
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>

#include <iostream>
#include <sstream>

namespace needle {
namespace cuda {

#define BASE_THREAD_NUM 256
#define TILE 16
typedef float scalar_t;
const size_t ELEM_SIZE = sizeof(scalar_t);

struct CudaArray {
  CudaArray(const size_t size) {
    hipError_t err = hipMalloc(&ptr, size * ELEM_SIZE);
    if (err != hipSuccess) throw std::runtime_error(hipGetErrorString(err));
    this->size = size;
  }
  ~CudaArray() { hipFree(ptr); }
  size_t ptr_as_int() { return (size_t)ptr; }
  
  scalar_t* ptr;
  size_t size;
};

struct CudaDims {
  dim3 block, grid;
};

CudaDims CudaOneDim(size_t size) {
  /**
   * Utility function to get cuda dimensions for 1D call
   */
  CudaDims dim;
  size_t num_blocks = (size + BASE_THREAD_NUM - 1) / BASE_THREAD_NUM;
  dim.block = dim3(BASE_THREAD_NUM, 1, 1);
  dim.grid = dim3(num_blocks, 1, 1);
  return dim;
}

CudaDims CudaTwoDim(size_t rows, size_t cols) {
  /**
   * Utility function to get cuda dimensions for 2D call
   */
  CudaDims dim;
  size_t num_row_blocks = (rows + TILE - 1) / TILE;
  size_t num_col_blocks = (cols + TILE - 1) / TILE;
  dim.block = dim3(TILE, TILE, 1);
  dim.grid = dim3(num_col_blocks, num_row_blocks, 1);
  return dim;
}

#define MAX_VEC_SIZE 8
template <typename T> 
struct CudaVec {
  uint32_t size;
  T data[MAX_VEC_SIZE];
};
template <typename T> 
CudaVec<T> VecToCuda(const std::vector<T>& x) {
  CudaVec<T> shape;
  if (x.size() > MAX_VEC_SIZE) throw std::runtime_error("Exceeded CUDA supported max dimesions");
  shape.size = x.size();
  for (size_t i = 0; i < x.size(); i++) {
    shape.data[i] = x[i];
  }
  return shape;
}

////////////////////////////////////////////////////////////////////////////////
// Fill call
////////////////////////////////////////////////////////////////////////////////

__global__ void FillKernel(scalar_t* out, scalar_t val, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = val;
}

void Fill(CudaArray* out, scalar_t val) {
  CudaDims dim = CudaOneDim(out->size);
  FillKernel<<<dim.grid, dim.block>>>(out->ptr, val, out->size);
}

////////////////////////////////////////////////////////////////////////////////
// Compact and setitem cals
////////////////////////////////////////////////////////////////////////////////

// Untility function to convert contiguous index i to memory location from strides


__device__ size_t GetNonCompactIndex(size_t gid, CudaVec<int> strides, CudaVec<uint32_t> compact_strides, size_t offset) {
  /**
   * Returns an index of a non-compact array with strides for the corresponding item (at location gid)
   * in the compact array out.
   * 
   * Args:
   *   gid: index of a compact array
   *   strides: vector of strides of a non-acompact array
   *   compact_strides: vector of compact strides of out array (derived based on the out shape)
   *   offset: offset of a non-acompact array
   */
  size_t index = offset;
  size_t remaining_elems = gid;
  for(size_t i=0; i<strides.size; i++) {
      index += (remaining_elems / compact_strides.data[i]) * strides.data[i];
      remaining_elems = remaining_elems % compact_strides.data[i];
  }
  return index;
}

__global__ void CompactKernel(const scalar_t* a, scalar_t* out, size_t size, CudaVec<uint32_t> shape,
                              CudaVec<int> strides, CudaVec<uint32_t> compact_strides, size_t offset) {
  /**
   * The CUDA kernel for the compact opeation.  This should effectively map a single entry in the 
   * non-compact input a, to the corresponding item (at location gid) in the compact array out.
   * 
   * Args:
   *   a: CUDA pointer to a array
   *   out: CUDA point to out array
   *   size: size of out array
   *   shape: vector of shapes of a and out arrays (of type CudaVec, for past passing to CUDA kernel)
   *   strides: vector of strides of out array
   *   compact_strides: vector of compact strides of out array (derived based on the out shape)
   *   offset: offset of out array
   */
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) {
    size_t src_index = GetNonCompactIndex(gid, strides, compact_strides, offset);
    out[gid] = a[src_index];
  }
}

std::vector<uint32_t> GetCompactStrides(std::vector<uint32_t>& shape) {
  /**
   * Returns compact strides based on array shape.
   * 
   * Args:
   *   shape: shapes of an array
   */
  std::vector<uint32_t> compact_strides(shape.size());
  uint32_t stride = 1;
  for(int i=shape.size()-1; i>=0; i--) {
    compact_strides[i] = stride;
    stride *= shape[i];
  } 
  return compact_strides;
}

void Compact(const CudaArray& a, CudaArray* out, std::vector<uint32_t> shape,
             std::vector<int> strides, size_t offset) {
  /**
   * Compact an array in memory.  Unlike the C++ version, in CUDA this will primarily call the 
   * relevant CUDA kernel.  In this case, we illustrate how you should set this up (i.e., we give 
   * you the code for this fuction, and also the prototype for the CompactKernel() function).  For
   * the functions after this, however, you'll need to define these kernels as you see fit to 
   * execute the underlying function.
   * 
   * Args:
   *   a: non-compact represntation of the array, given as input
   *   out: compact version of the array to be written
   *   shape: shapes of each dimension for a and out
   *   strides: strides of the *a* array (not out, which has compact strides)
   *   offset: offset of the *a* array (not out, which has zero offset, being compact)
   */
  std::vector<uint32_t> compact_strides = GetCompactStrides(shape);
  CudaDims dim = CudaOneDim(out->size);
  CompactKernel<<<dim.grid, dim.block>>>(a.ptr, out->ptr, out->size, VecToCuda<uint32_t>(shape),
                                         VecToCuda<int>(strides), VecToCuda<uint32_t>(compact_strides), offset);
}

__global__ void EwiseSetitemKernel(const scalar_t* a, scalar_t* out, size_t size, CudaVec<uint32_t> shape,
                                   CudaVec<int> strides, CudaVec<uint32_t> compact_strides, size_t offset) {
  /**
   * The CUDA kernel for the elementwise set item opeation. This will effectively set items from the 
   * compact *a* array  
   * 
   * Args:
   *   a: CUDA pointer to a array
   *   out: CUDA point to out array
   *   size: size of out array
   *   shape: vector of shapes of a and out arrays (of type CudaVec, for past passing to CUDA kernel)
   *   strides: vector of strides of out array
   *   compact_strides: vector of compact strides of out array (derived based on the out shape)
   *   offset: offset of out array
   */
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) {
    size_t dest_index = GetNonCompactIndex(gid, strides, compact_strides, offset);
    out[dest_index] = a[gid];
  }
}

void EwiseSetitem(const CudaArray& a, CudaArray* out, std::vector<uint32_t> shape,
                  std::vector<int> strides, size_t offset) {
  /**
   * Set items in a (non-compact) out array using CUDA.  You will most likely want to implement a
   * EwiseSetitemKernel() function, similar to those above, that will do the actual work.
   * 
   * Args:
   *   a: _compact_ array whose items will be written to out
   *   out: non-compact array whose items are to be written
   *   shape: shapes of each dimension for a and out
   *   strides: strides of the *out* array (not a, which has compact strides)
   *   offset: offset of the *out* array (not a, which has zero offset, being compact)
   */
  std::vector<uint32_t> compact_strides = GetCompactStrides(shape);
  CudaDims dim = CudaOneDim(out->size);
  EwiseSetitemKernel<<<dim.grid, dim.block>>>(a.ptr, out->ptr, a.size, VecToCuda<uint32_t>(shape),
                                              VecToCuda<int>(strides), VecToCuda<uint32_t>(compact_strides), offset);
}

__global__ void ScalarSetitemKernel(size_t size, scalar_t val, scalar_t* out, CudaVec<uint32_t> shape,
                                    CudaVec<int> strides, CudaVec<uint32_t> compact_strides, size_t offset) {
  /**
   * The CUDA kernel for the scalar set item opeation. This will effectively set items in out (non-compact)
   * array to value val.
   * 
   * Args:
   *   size: number of elements to write in out array (note that this will note be the same as
   *         out.size, because out is a non-compact subset array);  it _will_ be the same as the 
   *         product of items in shape, but covenient to just pass it here.
   *   val: scalar value to write to
   *   out: non-compact array whose items are to be written
   *   shape: shapes of each dimension of out
   *   strides: strides of the out array
   *   offset: offset of the out array
   */
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) {
    size_t dest_index = GetNonCompactIndex(gid, strides, compact_strides, offset);
    out[dest_index] = val;
  }
}

void ScalarSetitem(size_t size, scalar_t val, CudaArray* out, std::vector<uint32_t> shape,
                   std::vector<int> strides, size_t offset) {
  /**
   * Set items is a (non-compact) array
   * 
   * Args:
   *   size: number of elements to write in out array (note that this will note be the same as
   *         out.size, because out is a non-compact subset array);  it _will_ be the same as the 
   *         product of items in shape, but covenient to just pass it here.
   *   val: scalar value to write to
   *   out: non-compact array whose items are to be written
   *   shape: shapes of each dimension of out
   *   strides: strides of the out array
   *   offset: offset of the out array
   */
  std::vector<uint32_t> compact_strides = GetCompactStrides(shape);
  CudaDims dim = CudaOneDim(out->size);
  ScalarSetitemKernel<<<dim.grid, dim.block>>>(size, val, out->ptr, VecToCuda(shape), VecToCuda(strides), 
                                              VecToCuda(compact_strides), offset);
}

////////////////////////////////////////////////////////////////////////////////
// Elementwise and scalar operations
////////////////////////////////////////////////////////////////////////////////

__global__ void EwiseAddKernel(const scalar_t* a, const scalar_t* b, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = a[gid] + b[gid];
}

void EwiseAdd(const CudaArray& a, const CudaArray& b, CudaArray* out) {
  /**
   * Add together two CUDA array
   */
  CudaDims dim = CudaOneDim(out->size);
  EwiseAddKernel<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, out->size);
}

__global__ void ScalarAddKernel(const scalar_t* a, scalar_t val, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = a[gid] + val;
}

void ScalarAdd(const CudaArray& a, scalar_t val, CudaArray* out) {
  /**
   * Add together a CUDA array and a scalar value.
   */
  CudaDims dim = CudaOneDim(out->size);
  ScalarAddKernel<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size);
}

/**
 * In the code the follows, use the above template to create analogous elementise
 * and and scalar operators for the following functions.  See the numpy backend for
 * examples of how they should work.
 *   - EwiseMul, ScalarMul
 *   - EwiseDiv, ScalarDiv
 *   - ScalarPower
 *   - EwiseMaximum, ScalarMaximum
 *   - EwiseEq, ScalarEq
 *   - EwiseGe, ScalarGe
 *   - EwiseLog
 *   - EwiseExp
 *   - EwiseTanh
 *
 * If you implement all these naively, there will be a lot of repeated code, so
 * you are welcome (but not required), to use macros or templates to define these
 * functions (however you want to do so, as long as the functions match the proper)
 * signatures above.
 */

/// BEGIN YOUR SOLUTION
__global__ void EwiseMulKernel(const scalar_t* a, const scalar_t* b, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = a[gid] * b[gid];
}

void EwiseMul(const CudaArray& a, const CudaArray& b, CudaArray* out) {
  /**
   * Multiply together two CUDA array
   */
  CudaDims dim = CudaOneDim(out->size);
  EwiseMulKernel<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, out->size);
}

__global__ void ScalarMulKernel(const scalar_t* a, scalar_t val, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = a[gid] * val;
}

void ScalarMul(const CudaArray& a, scalar_t val, CudaArray* out) {
  /**
   * Multiply together a CUDA array and a scalar value.
   */
  CudaDims dim = CudaOneDim(out->size);
  ScalarMulKernel<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size);
}

__global__ void EwiseDivKernel(const scalar_t* a, const scalar_t* b, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = a[gid] / b[gid];
}

void EwiseDiv(const CudaArray& a, const CudaArray& b, CudaArray* out) {
  /**
   * Divide a CUDA array by an another one.
   */
  CudaDims dim = CudaOneDim(out->size);
  EwiseDivKernel<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, out->size);
}

__global__ void ScalarDivKernel(const scalar_t* a, scalar_t val, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = a[gid] / val;
}

void ScalarDiv(const CudaArray& a, scalar_t val, CudaArray* out) {
  /**
   * Divide a CUDA array by a scalar value.
   */
  CudaDims dim = CudaOneDim(out->size);
  ScalarDivKernel<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size);
}

__global__ void ScalarPowerKernel(const scalar_t* a, scalar_t val, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = pow(a[gid], val);
}

void ScalarPower(const CudaArray& a, scalar_t val, CudaArray* out) {
  /**
   * Power a CUDA array to a scalar value.
   */
  CudaDims dim = CudaOneDim(out->size);
  ScalarPowerKernel<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size);
}

__global__ void EwiseMaximumKernel(const scalar_t* a, const scalar_t* b, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = max(a[gid], b[gid]);
}

void EwiseMaximum(const CudaArray& a, const CudaArray& b, CudaArray* out) {
  /**
   * Find elementwise maximum of two CUDA arrays
   */
  CudaDims dim = CudaOneDim(out->size);
  EwiseMaximumKernel<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, out->size);
}

__global__ void ScalarMaximumKernel(const scalar_t* a, scalar_t val, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = max(a[gid], val);
}

void ScalarMaximum(const CudaArray& a, scalar_t val, CudaArray* out) {
  /**
   * Find maximum of a CUDA array and a scalar value.
   */
  CudaDims dim = CudaOneDim(out->size);
  ScalarMaximumKernel<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size);
}

__global__ void EwiseEqKernel(const scalar_t* a, const scalar_t* b, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = (a[gid] == b[gid]);
}

void EwiseEq(const CudaArray& a, const CudaArray& b, CudaArray* out) {
  /**
   * Find if two CUDA arrays are elementwise equal.
   */
  CudaDims dim = CudaOneDim(out->size);
  EwiseEqKernel<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, out->size);
}

__global__ void ScalarEqKernel(const scalar_t* a, scalar_t val, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = (a[gid] == val);
}

void ScalarEq(const CudaArray& a, scalar_t val, CudaArray* out) {
  /**
   * Find if a CUDA array is elementwise equal to a scalar.
   */
  CudaDims dim = CudaOneDim(out->size);
  ScalarEqKernel<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size);
}

__global__ void EwiseGeKernel(const scalar_t* a, const scalar_t* b, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = (a[gid] >= b[gid]);
}

void EwiseGe(const CudaArray& a, const CudaArray& b, CudaArray* out) {
  /**
   * Find if two CUDA arrays are elementwise greater or equal.
   */
  CudaDims dim = CudaOneDim(out->size);
  EwiseGeKernel<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, out->size);
}

__global__ void ScalarGeKernel(const scalar_t* a, scalar_t val, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = (a[gid] >= val);
}

void ScalarGe(const CudaArray& a, scalar_t val, CudaArray* out) {
  /**
   * Find if a CUDA array is elementwise greater or equal to a scalar.
   */
  CudaDims dim = CudaOneDim(out->size);
  ScalarGeKernel<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size);
}

__global__ void EwiseLogKernel(const scalar_t* a, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = log(a[gid]);
}

void EwiseLog(const CudaArray& a, CudaArray* out) {
  /**
   * Find elementwise log of a CUDA array.
   */
  CudaDims dim = CudaOneDim(out->size);
  EwiseLogKernel<<<dim.grid, dim.block>>>(a.ptr, out->ptr, out->size);
}

__global__ void EwiseExpKernel(const scalar_t* a, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = exp(a[gid]);
}

void EwiseExp(const CudaArray& a, CudaArray* out) {
  /**
   * Find elementwise exp of a CUDA array.
   */
  CudaDims dim = CudaOneDim(out->size);
  EwiseExpKernel<<<dim.grid, dim.block>>>(a.ptr, out->ptr, out->size);
}

__global__ void EwiseTanhKernel(const scalar_t* a, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = tanh(a[gid]);
}

void EwiseTanh(const CudaArray& a, CudaArray* out) {
  /**
   * Find elementwise tanh of a CUDA array.
   */
  CudaDims dim = CudaOneDim(out->size);
  EwiseTanhKernel<<<dim.grid, dim.block>>>(a.ptr, out->ptr, out->size);
}

/// END YOUR SOLUTION

////////////////////////////////////////////////////////////////////////////////
// Matmul
////////////////////////////////////////////////////////////////////////////////
__global__ void MatmulKernelShared(const scalar_t* a, const scalar_t* b, scalar_t* out, size_t M, size_t N, size_t P) {
  size_t row = blockIdx.y * blockDim.y + threadIdx.y;
  size_t col = blockIdx.x * blockDim.x + threadIdx.x;
  __shared__ scalar_t shared_a[TILE][TILE];
  __shared__ scalar_t shared_b[TILE][TILE];
  size_t tile_r = threadIdx.y;
  size_t tile_c = threadIdx.x;
  scalar_t out_val = 0.;
  
  for(size_t k=0; k<(N+TILE-1)/TILE; k++) {
    shared_a[tile_r][tile_c] = (row<M && (k*TILE+tile_c)<N)? a[row*N + k*TILE+tile_c]: 0.;
    shared_b[tile_r][tile_c] = ((k*TILE+tile_r)<N && col<P)? b[(k*TILE+tile_r)*P + col]: 0.;
    __syncthreads();

    for(int m=0; m<TILE; m++) {
      out_val += shared_a[tile_r][m] * shared_b[m][tile_c];
    }
    __syncthreads();
  }
  if((row < M) && (col < P)) {
    out[row*P + col] = out_val;
  }
}

__global__ void MatmulKernelRegisters(const scalar_t* a, const scalar_t* b, scalar_t* out, size_t M, size_t N, size_t P) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  size_t start_row = (gid / ((P+TILE-1) / TILE)) * TILE;
  size_t start_col = (gid % ((P+TILE-1) / TILE)) * TILE;
  scalar_t rc[TILE][TILE] = {0.};
  scalar_t ra[TILE], rb[TILE];
  
  for(size_t k=0; k<N; k++) {
    for(int i=0; i<TILE; i++) {
      ra[i] = ((start_row+i) < M)? a[(start_row + i)*N + k]: 0.;
      rb[i] = ((start_col+i) < P)? b[k*P + (start_col+i)]: 0.;
    }
    for(int i=0; i<TILE; i++) {
      for(int j=0; j<TILE; j++) {
        rc[i][j] += (ra[i] * rb[j]);
      }
    }
  }
  
  for(int i=0; i<TILE; i++) {
    for(int j=0; j<TILE; j++) {
      if(((start_row+i) < M) && ((start_col+j) < P)) {
        out[(start_row+i)*P + (start_col+j)] = rc[i][j];
      }
    }
  }
}

void Matmul(const CudaArray& a, const CudaArray& b, CudaArray* out, uint32_t M, uint32_t N,
            uint32_t P) {
  /**
   * Multiply two (compact) matrices into an output (also comapct) matrix.  You will want to look
   * at the lecture and notes on GPU-based linear algebra to see how to do this.  Since ultimately
   * mugrade is just evaluating correctness, you _can_ implement a version that simply parallelizes
   * over (i,j) entries in the output array.  However, to really get the full benefit of this
   * problem, we would encourage you to use cooperative fetching, shared memory register tiling, 
   * and other ideas covered in the class notes.  Note that unlike the tiled matmul function in
   * the CPU backend, here you should implement a single function that works across all size
   * matrices, whether or not they are a multiple of a tile size.  As with previous CUDA
   * implementations, this function here will largely just set up the kernel call, and you should
   * implement the logic in a separate MatmulKernel() call.
   * 
   *
   * Args:
   *   a: compact 2D array of size m x n
   *   b: comapct 2D array of size n x p
   *   out: compact 2D array of size m x p to write the output to
   *   M: rows of a / out
   *   N: columns of a / rows of b
   *   P: columns of b / out
   */
  // CudaDims dim = CudaOneDim(((M + TILE - 1) / TILE) * ((P + TILE - 1) / TILE));
  // MatmulKernelRegisters<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, M, N, P); 
  CudaDims dim = CudaTwoDim(M, P);
  MatmulKernelShared<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, M, N, P);
}

////////////////////////////////////////////////////////////////////////////////
// Max and sum reductions
////////////////////////////////////////////////////////////////////////////////

__global__ void ReduceMaxKernel(const scalar_t* a, scalar_t* out, size_t size, size_t reduce_size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) {
    scalar_t max_val = 0.;
    for(size_t j=0; j<reduce_size; j++) {
      if (j == 0)
        max_val = a[gid*reduce_size + j];
      else
        max_val = max(max_val, a[gid*reduce_size + j]);
    }
    out[gid] = max_val;
  }
}

void ReduceMax(const CudaArray& a, CudaArray* out, size_t reduce_size) {
  /**
   * Reduce by taking maximum over `reduce_size` contiguous blocks.  Even though it is inefficient,
   * for simplicity you can perform each reduction in a single CUDA thread.
   * 
   * Args:
   *   a: compact array of size a.size = out.size * reduce_size to reduce over
   *   out: compact array to write into
   *   redice_size: size of the dimension to reduce over
   */
  CudaDims dim = CudaOneDim(out->size);
  ReduceMaxKernel<<<dim.grid, dim.block>>>(a.ptr, out->ptr, out->size, reduce_size);
}

__global__ void ReduceSumKernel(const scalar_t* a, scalar_t* out, size_t size, size_t reduce_size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) {
    scalar_t sum_val = 0.;
    for(size_t j=0; j<reduce_size; j++) {
        sum_val += a[gid*reduce_size + j];
    }
    out[gid] = sum_val;
  }
}

void ReduceSum(const CudaArray& a, CudaArray* out, size_t reduce_size) {
  /**
   * Reduce by taking summation over `reduce_size` contiguous blocks.  Again, for simplicity you 
   * can perform each reduction in a single CUDA thread.
   * 
   * Args:
   *   a: compact array of size a.size = out.size * reduce_size to reduce over
   *   out: compact array to write into
   *   redice_size: size of the dimension to reduce over
   */
  CudaDims dim = CudaOneDim(out->size);
  ReduceSumKernel<<<dim.grid, dim.block>>>(a.ptr, out->ptr, out->size, reduce_size);
}

}  // namespace cuda
}  // namespace needle

PYBIND11_MODULE(ndarray_backend_cuda, m) {
  namespace py = pybind11;
  using namespace needle;
  using namespace cuda;

  m.attr("__device_name__") = "cuda";
  m.attr("__tile_size__") = TILE;

  py::class_<CudaArray>(m, "Array")
      .def(py::init<size_t>(), py::return_value_policy::take_ownership)
      .def_readonly("size", &CudaArray::size)
      .def("ptr", &CudaArray::ptr_as_int);

  // return numpy array, copying from CPU
  m.def("to_numpy", [](const CudaArray& a, std::vector<size_t> shape, std::vector<size_t> strides,
                       size_t offset) {
    std::vector<size_t> numpy_strides = strides;
    std::transform(numpy_strides.begin(), numpy_strides.end(), numpy_strides.begin(),
                   [](size_t& c) { return c * ELEM_SIZE; });

    // copy memory to host
    scalar_t* host_ptr = (scalar_t*)std::malloc(a.size * ELEM_SIZE);
    if (host_ptr == 0) throw std::bad_alloc();
    hipError_t err = hipMemcpy(host_ptr, a.ptr, a.size * ELEM_SIZE, hipMemcpyDeviceToHost);
    if (err != hipSuccess) throw std::runtime_error(hipGetErrorString(err));

    // return numpy array
    py::capsule deallocate_buffer(host_ptr, [](void* p) { free(p); });
    return py::array_t<scalar_t>(shape, numpy_strides, host_ptr + offset, deallocate_buffer);
  });

  // copy numpy array to GPU
  m.def("from_numpy", [](py::array_t<scalar_t> a, CudaArray* out) {
    hipError_t err =
        hipMemcpy(out->ptr, a.request().ptr, out->size * ELEM_SIZE, hipMemcpyHostToDevice);
    if (err != hipSuccess) throw std::runtime_error(hipGetErrorString(err));
  });

  m.def("fill", Fill);
  m.def("compact", Compact);
  m.def("ewise_setitem", EwiseSetitem);
  m.def("scalar_setitem", ScalarSetitem);
  m.def("ewise_add", EwiseAdd);
  m.def("scalar_add", ScalarAdd);

  m.def("ewise_mul", EwiseMul);
  m.def("scalar_mul", ScalarMul);
  m.def("ewise_div", EwiseDiv);
  m.def("scalar_div", ScalarDiv);
  m.def("scalar_power", ScalarPower);

  m.def("ewise_maximum", EwiseMaximum);
  m.def("scalar_maximum", ScalarMaximum);
  m.def("ewise_eq", EwiseEq);
  m.def("scalar_eq", ScalarEq);
  m.def("ewise_ge", EwiseGe);
  m.def("scalar_ge", ScalarGe);

  m.def("ewise_log", EwiseLog);
  m.def("ewise_exp", EwiseExp);
  m.def("ewise_tanh", EwiseTanh);

  m.def("matmul", Matmul);

  m.def("reduce_max", ReduceMax);
  m.def("reduce_sum", ReduceSum);
}
